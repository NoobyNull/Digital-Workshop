#include "hip/hip_runtime.h"
/**
 * CUDA kernel for GPU-accelerated STL triangle processing.
 *
 * This kernel processes raw STL binary data to extract triangle geometry
 * and compute normals, optimized for parallel execution on NVIDIA GPUs.
 *
 * Performance targets:
 * - 10-20x speedup over CPU processing for large models
 * - Memory-efficient processing with minimal host-device transfers
 * - Support for up to 10M triangles per kernel launch
 */

#include <hip/hip_runtime.h>
#include <>
#include <math.h>

// STL triangle structure (50 bytes total)
struct STLTriangle {
    float normal[3];      // 12 bytes
    float vertices[9];    // 36 bytes (3 vertices × 3 floats)
    unsigned short attr;  // 2 bytes (attribute byte count)
};

// Output vertex structure for VTK integration
struct ProcessedVertex {
    float position[3];
    float normal[3];
};

/**
 * Kernel to process STL triangles and extract vertex/normal data.
 *
 * Each thread processes one triangle from the raw STL data.
 * Outputs are stored in separate vertex and normal arrays for efficient
 * memory access patterns in downstream processing.
 *
 * @param stl_data Raw STL triangle data (50 bytes per triangle)
 * @param vertex_output Output vertex array (9 floats per triangle)
 * @param normal_output Output normal array (3 floats per triangle, repeated 3x)
 * @param triangle_count Total number of triangles to process
 */
__global__ void process_stl_triangles_kernel(
    const unsigned char* stl_data,
    float* vertex_output,
    float* normal_output,
    const unsigned int triangle_count
) {
    // Calculate global triangle index
    const unsigned int triangle_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check
    if (triangle_idx >= triangle_count) {
        return;
    }

    // Calculate input data offset (50 bytes per triangle)
    const size_t data_offset = triangle_idx * 50;

    // Extract normal vector (first 12 bytes, little-endian floats)
    const float* normal_ptr = reinterpret_cast<const float*>(&stl_data[data_offset]);
    float nx = normal_ptr[0];
    float ny = normal_ptr[1];
    float nz = normal_ptr[2];

    // Extract vertex data (next 36 bytes, 9 floats)
    const float* vertex_ptr = reinterpret_cast<const float*>(&stl_data[data_offset + 12]);
    float v1x = vertex_ptr[0], v1y = vertex_ptr[1], v1z = vertex_ptr[2];
    float v2x = vertex_ptr[3], v2y = vertex_ptr[4], v2z = vertex_ptr[5];
    float v3x = vertex_ptr[6], v3y = vertex_ptr[7], v3z = vertex_ptr[8];

    // Calculate output offsets
    const size_t vertex_out_offset = triangle_idx * 9;  // 9 floats per triangle
    const size_t normal_out_offset = triangle_idx * 9;  // 3 normals × 3 floats each

    // Store vertices (interleaved: v1x,v1y,v1z,v2x,v2y,v2z,v3x,v3y,v3z)
    vertex_output[vertex_out_offset + 0] = v1x;
    vertex_output[vertex_out_offset + 1] = v1y;
    vertex_output[vertex_out_offset + 2] = v1z;
    vertex_output[vertex_out_offset + 3] = v2x;
    vertex_output[vertex_out_offset + 4] = v2y;
    vertex_output[vertex_out_offset + 5] = v2z;
    vertex_output[vertex_out_offset + 6] = v3x;
    vertex_output[vertex_out_offset + 7] = v3y;
    vertex_output[vertex_out_offset + 8] = v3z;

    // Store normals (repeated 3 times, one per vertex)
    #pragma unroll
    for (int i = 0; i < 3; ++i) {
        const size_t normal_idx = normal_out_offset + i * 3;
        normal_output[normal_idx + 0] = nx;
        normal_output[normal_idx + 1] = ny;
        normal_output[normal_idx + 2] = nz;
    }
}

/**
 * Kernel to compute bounding box from vertex data.
 *
 * Processes vertex array to find min/max bounds using parallel reduction.
 * Each block computes partial min/max, final reduction done on host.
 *
 * @param vertices Input vertex array (x,y,z,x,y,z,... format)
 * @param vertex_count Total number of vertices (3 per triangle)
 * @param partial_mins Output array for block-level min values
 * @param partial_maxs Output array for block-level max values
 */
__global__ void compute_bounds_kernel(
    const float* vertices,
    const unsigned int vertex_count,
    float* partial_mins,
    float* partial_maxs
) {
    // Shared memory for block-level reduction
    __shared__ float shared_min[3];
    __shared__ float shared_max[3];

    // Initialize shared memory
    if (threadIdx.x == 0) {
        shared_min[0] = INFINITY;
        shared_min[1] = INFINITY;
        shared_min[2] = INFINITY;
        shared_max[0] = -INFINITY;
        shared_max[1] = -INFINITY;
        shared_max[2] = -INFINITY;
    }
    __syncthreads();

    // Each thread processes multiple vertices for better occupancy
    const unsigned int stride = blockDim.x * gridDim.x;
    float local_min[3] = {INFINITY, INFINITY, INFINITY};
    float local_max[3] = {-INFINITY, -INFINITY, -INFINITY};

    for (unsigned int i = threadIdx.x + blockIdx.x * blockDim.x; i < vertex_count; i += stride) {
        const unsigned int base_idx = i * 3;
        const float x = vertices[base_idx + 0];
        const float y = vertices[base_idx + 1];
        const float z = vertices[base_idx + 2];

        local_min[0] = fminf(local_min[0], x);
        local_min[1] = fminf(local_min[1], y);
        local_min[2] = fminf(local_min[2], z);

        local_max[0] = fmaxf(local_max[0], x);
        local_max[1] = fmaxf(local_max[1], y);
        local_max[2] = fmaxf(local_max[2], z);
    }

    // Reduce within block
    atomicMinFloat(&shared_min[0], local_min[0]);
    atomicMinFloat(&shared_min[1], local_min[1]);
    atomicMinFloat(&shared_min[2], local_min[2]);

    atomicMaxFloat(&shared_max[0], local_max[0]);
    atomicMaxFloat(&shared_max[1], local_max[1]);
    atomicMaxFloat(&shared_max[2], local_max[2]);

    __syncthreads();

    // Write block results to global memory
    if (threadIdx.x == 0) {
        const unsigned int block_idx = blockIdx.x * 3;
        partial_mins[block_idx + 0] = shared_min[0];
        partial_mins[block_idx + 1] = shared_min[1];
        partial_mins[block_idx + 2] = shared_min[2];

        partial_maxs[block_idx + 0] = shared_max[0];
        partial_maxs[block_idx + 1] = shared_max[1];
        partial_maxs[block_idx + 2] = shared_max[2];
    }
}

/**
 * Kernel to validate STL triangle data integrity.
 *
 * Checks for NaN/inf values and degenerate triangles.
 * Outputs validation flags for each triangle.
 *
 * @param vertices Input vertex array
 * @param triangle_count Number of triangles to validate
 * @param validation_flags Output flags (0=valid, 1=invalid)
 */
__global__ void validate_triangles_kernel(
    const float* vertices,
    const unsigned int triangle_count,
    unsigned char* validation_flags
) {
    const unsigned int triangle_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (triangle_idx >= triangle_count) {
        return;
    }

    const size_t vertex_offset = triangle_idx * 9;
    unsigned char is_valid = 1;

    // Check for NaN or infinite values
    #pragma unroll
    for (int i = 0; i < 9; ++i) {
        const float val = vertices[vertex_offset + i];
        if (isnan(val) || isinf(val)) {
            is_valid = 0;
            break;
        }
    }

    // Check for degenerate triangles (zero area)
    if (is_valid) {
        const float* v1 = &vertices[vertex_offset + 0];
        const float* v2 = &vertices[vertex_offset + 3];
        const float* v3 = &vertices[vertex_offset + 6];

        // Compute cross product to check area
        const float ux = v2[0] - v1[0], uy = v2[1] - v1[1], uz = v2[2] - v1[2];
        const float vx = v3[0] - v1[0], vy = v3[1] - v1[1], vz = v3[2] - v1[2];

        const float cross_x = uy * vz - uz * vy;
        const float cross_y = uz * vx - ux * vz;
        const float cross_z = ux * vy - uy * vx;

        const float area_squared = cross_x*cross_x + cross_y*cross_y + cross_z*cross_z;

        if (area_squared < 1e-12f) {  // Very small area threshold
            is_valid = 0;
        }
    }

    validation_flags[triangle_idx] = is_valid;
}

// Helper functions for atomic operations on floats (CUDA < 11.0 compatibility)
__device__ float atomicMinFloat(float* addr, float val) {
    return atomicMin((int*)addr, __float_as_int(val));
}

__device__ float atomicMaxFloat(float* addr, float val) {
    return atomicMax((int*)addr, __float_as_int(val));
}